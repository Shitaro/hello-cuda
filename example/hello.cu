
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void cuda_hello(void) {
    printf("Hello World from GPU!\n");
}

int main(void)
{
    cuda_hello<<<1,1000>>>();
    hipDeviceSynchronize();
    return 0;
}
