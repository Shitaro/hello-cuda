
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

/* device function */
/* device function processed by NVIDIA compiler */
__global__ // Runs on the device, called from host code
void cuda_hello(void) {
    printf("Hello World from GPU!\n");
}

int main(void)
{
    std::cout << "Hello World from Host" << std::endl;

    cuda_hello<<<1,1000>>>(); // kernel launch: triple angle bracets mark a call from host code to device code

    hipDeviceSynchronize();
    return 0;
}
