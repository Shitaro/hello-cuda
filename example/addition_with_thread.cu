/* For Device Code */

#include <hip/hip_runtime.h>
#include <stdio.h>

/* For Host Code */
#include <iostream>
#include <random>

/* device function */
/* device function processed by NVIDIA compiler */
__global__ // Runs on the device, called from host code
void add(int* a, int* b, int* c) {
    /* a, b, and c must point to device memory */
    /* threadIdx.x: to access thread index */
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int* a, int N) {
    std::mt19937 mt{ std::random_device{}() };
    std::uniform_int_distribution<int> dist(0, 255);
    for (int i = 0; i < N; i++) {
        a[i] = dist(mt);
    }
}

/* host function */
int main(void) {
    static constexpr int N = 512;
    int *a, *b, *c;    // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c
    int size = N * sizeof(int);

    /* allocate space for device copies of a, b, c */
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    /* allocate space for host copies of a, b, c */
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    /* setup input values */
    random_ints(a, N);
    random_ints(b, N);

    /* copy inputs to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    /* launch add() kernel on GPU with N threads */
    /* thread: block can be split into parallel threads */
    add<<<1,N>>>(d_a, d_b, d_c); // kernel launch: triple angle bracets mark a call from host code to device code

    /* copy result back to host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        std::cout << a[i] << "+" << b[i] << "=" << c[i] << std::endl;
    }

    /* cleanup */
    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
