#include "exec_add_vector.h"

#include <iostream>
#include <cstdint>
#include <random>

#include <mycuda/add_vector.cu>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

template <typename T>
void random_float(T* a, std::int32_t N) {
    std::mt19937 mt(std::random_device{}());
    std::uniform_real_distribution<float> dist;

    for (std::int32_t idx = 0; idx < N; idx++) {
        a[idx] = dist(mt);
    }
}

template <typename T>
void exec_add_vector(std::int32_t N) {
    T *host_a, *host_b, *host_c;
    T *dev_a, *dev_b, *dev_c;
    std::int32_t size = N * sizeof(T);

    hipHostMalloc((void**)&host_a, size);
    hipHostMalloc((void**)&host_b, size);
    hipHostMalloc((void**)&host_c, size);

    random_float(host_a, N);
    random_float(host_b, N);

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    hipMemcpy(dev_a, host_a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, size, hipMemcpyHostToDevice);

    mycuda::add_vector<<<1, N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(host_c, dev_c, size, hipMemcpyDeviceToHost);

    std::cout << "host_a[" << N - 1 << "]: " << host_a[N - 1] << ", "
              << "host_b[" << N - 1 << "]: " << host_b[N - 1] << ", "
              << "host_c[" << N - 1 << "]: " << host_c[N - 1] << std::endl;

    std::cout << host_a[N - 1] << " + " << host_b[N - 1] << " = " << host_c[N - 1] << std::endl;

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

template void exec_add_vector<float>(std::int32_t N);
template void exec_add_vector<double>(std::int32_t N);